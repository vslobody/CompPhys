/**
 * @file   : 024stbsv_b.cu
 * @brief  : cublasStbsv - solve the triangular banded linear system  
 * uses CUDA Unified Memory (Management); bare minimum
 * @author : Ernest Yeung	ernestyalumni@gmail.com
 * @date   : 20170417
 * @ref    :  cf. https://developer.nvidia.com/sites/default/files/akamai/cuda/files/Misc/mygpu.pdf
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
// COMPILATION TIP:
// nvcc -std=c++11 -arch='sm_52' 024stbsv_b.cu -lcublas -o 024stbsv_b.exe

#include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"

constexpr const int n =6;			// number of rows and columns of a 
constexpr const int k =1;				// number of subdiagonals

__device__ __managed__ float A[n*n];  	// nxn matrix A on CUDA Unified (managed) memory 
__device__ __managed__ float b[n];	// n-vector b on CUDA Unified (managed) memory 
__device__ __managed__ float x[n];  // n-vector x on CUDA Unified (managed) memory

int main(void) {
	hipError_t cudaStat;					// cudaMalloc status
	hipblasStatus_t stat;					// CUBLAS functions status
	hipblasHandle_t handle;					// CUBLAS context
	int i,j;							// lower triangle of a:
	
	// main diagonal and subdiagonals of A in rows:
	int ind=11;
	// main diagonal: 11, 12,13,14,15,16 in row 0
	for (i=0; i<n; i++) {
		A[i*n]=(float)ind++;  
	}
	
	// first subdiagonal: 17, 18, 19, 20, 21 in row 1
	for (i=0;i<n-1;i++) {
		A[i*n+1]=(float)ind++; 
	}
		
	for (i=0; i<n;i++) {
		b[i]=1.0f; 
	}					// b={1,1,1,1,1,1}^T

	for (j=0;j<n;j++){
		for (i=0;i<n;i++){
			std::cout << A[i + n*j] << " "; 
	} 
		std::cout << std::endl; }

	for (i=0;i<n; i++) {
		std::cout << b[i] << " "; } std::cout << std::endl;

		
	stat = hipblasCreate(&handle);		// initialize CUBLAS context

	/* print out */
	for (j=0;j<n;j++){
		for (i=0;i<n;i++){
			std::cout << A[i + n*j] << " "; 
	} 
		std::cout << std::endl; }
	/* END of print out */


	// solve a triangular banded linear system: Ax=b;
	// the solution x overwrite the right hand side (RHS) b;
	// A - nxn banded lower triangular matrix; b - n-vector

	stat=hipblasStbsv(handle,HIPBLAS_FILL_MODE_LOWER,HIPBLAS_OP_N,
							HIPBLAS_DIAG_NON_UNIT, n,k, A,n, b,1);

	// for some reason, if cublasGetVector isn't included, Bus error (core dumped) isn't obtained
	stat = hipblasGetVector(n,sizeof(float),b,1,x,1);

	// print the solution
	std::cout << "solution : " << std::endl; 	// print x after Stbsv
	for(j=0;j<n;j++) 
	{
		std::cout << b[j] << std::endl; }
	for(j=0;j<n;j++) 
	{
		std::cout << x[j] << std::endl; }



	hipblasDestroy(handle);	// destroy CUBLAS context	
	
	hipDeviceReset();
	return EXIT_SUCCESS;

}
